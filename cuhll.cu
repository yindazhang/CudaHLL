#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <random>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// from https://github.com/jarro2783/cxxopts
#include "cxxopts.hpp"

#define cudaCheck(err) (cudaErrorCheck(err, __FILE__, __LINE__))
#define cublasCheck(err) (cublasErrorCheck(err, __FILE__, __LINE__))
#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

#define BLOCK_SIZE

enum Algo
{
    cpu = 0,
    basic,
    opt,
    numAlgos
};

const char *algo2str(Algo a)
{
    switch (a)
    {
    case cpu:
        return "cpu";
    case basic:
        return "basic";
    case opt:
        return "opt";
    default:
        return "INVALID";
    }
}

void cudaErrorCheck(hipError_t error, const char *file, int line);
void cublasErrorCheck(hipblasStatus_t status, const char *file, int line);
void randomize_vector(uint32_t *mat, int N, double skewness);
bool verify_hll(uint8_t *expected, uint8_t *actual, int M);
void runAlgo(Algo algo, int N, int M, uint32_t *A, uint8_t *C, uint32_t *dA, uint8_t *dC);
void runCpu(int N, int M, uint32_t *A, uint8_t *C);

const std::string errLogFile = "hllValidationFailure.txt";

// NB: must use a single generator to avoid duplicates
std::mt19937 distribution;

int main(int argc, char **argv)
{
    // command-line flags
    cxxopts::Options options("hll.cu", "CUDA HLL kernels");
    options.add_options()("size", "dataset size (2^N)", cxxopts::value<uint8_t>()->default_value("20"))
        ("skew", "skewness of the dataset", cxxopts::value<double>()->default_value("0")) //TODO
        ("bconfig", "b in the configuration of HLL", cxxopts::value<uint8_t>()->default_value("10"))
        ("reps", "repeat HLL this many times", cxxopts::value<uint16_t>()->default_value("1"))
        ("algo", "HLL algorithm to use, a number in [0,4], 0 is cuBLAS", cxxopts::value<uint16_t>()->default_value("0"))
        ("validate", "Validate output against cuBLAS", cxxopts::value<bool>()->default_value("true"))
        ("rngseed", "PRNG seed", cxxopts::value<uint>()->default_value("2"))     
        ("h,help", "Print usage");

    auto clFlags = options.parse(argc, argv);
    if (clFlags.count("help"))
    {
        std::cout << options.help() << std::endl;
        exit(0);
    }

    uint32_t SIZE = clFlags["size"].as<uint8_t>();
    if (SIZE > 25)
    {
        std::cout << "--size must be smaller than 25" << std::endl;
        exit(EXIT_FAILURE);
    }
    SIZE = (1 << SIZE);
    double skewness = clFlags["skew"].as<double>();

    uint8_t HLLB = clFlags["bconfig"].as<uint8_t>();
    if (HLLB < 4 || HLLB > 25)
    {
        std::cout << "--b in HLL must be in the range [4,25]" << std::endl;
        exit(EXIT_FAILURE);
    }
    uint32_t HLLM = (1 << HLLB);


    const uint16_t REPS = clFlags["reps"].as<uint16_t>();
    const Algo ALGO = static_cast<Algo>(clFlags["algo"].as<uint16_t>());
    if (ALGO >= numAlgos)
    {
        printf("Invalid algorithm: %d\n", ALGO);
        exit(EXIT_FAILURE);
    }

    const bool VALIDATE = clFlags["validate"].as<bool>();
    const uint SEED = clFlags["rngseed"].as<uint>();
    distribution.seed(SEED);
    printf("Multiplying two %u x %u matrices with %u trials using %s algorithm\n", SIZE, SIZE, REPS, algo2str(ALGO));

    cudaCheck(hipSetDevice(0));

    // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
    // publishing event tasks in the target stream
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    uint32_t *A = nullptr, *dA = nullptr;
    uint8_t *C = nullptr, *C_ref = nullptr, *dC = nullptr;

    A = (uint32_t *)malloc(sizeof(uint32_t) * SIZE);
    C = (uint8_t *)malloc(sizeof(uint8_t) * HLLM);
    C_ref = (uint8_t *)malloc(sizeof(uint8_t) * HLLM);

    randomize_vector(A, SIZE, skewness);
    memset(C, 0, sizeof(uint8_t) * HLLM);
    memset(C_ref, 0, sizeof(uint8_t) * HLLM);

    cudaCheck(hipMalloc((void **)&dA, sizeof(uint32_t) * SIZE));
    cudaCheck(hipMalloc((void **)&dC, sizeof(uint8_t) * HLLM));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(uint8_t) * HLLM));

    cudaCheck(hipMemcpy(dA, A, sizeof(uint32_t) * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(uint8_t) * HLLM, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(uint8_t) * HLLM, hipMemcpyHostToDevice));

    printf("size: %u, m in HLL: %u\n", SIZE, HLLM);

    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (!VALIDATE)
    {
        printf("disabled validation\n");
    }
    else
    {
        // run cublas to get correct answer in dC_ref
        runCpu(SIZE, HLLM, A, C_ref);

        // run user's algorithm, filling in dC
        runAlgo(ALGO, SIZE, HLLM, A, C, dA, dC);

        cudaCheck(hipDeviceSynchronize());

        // copy both results back to host
        hipMemcpy(C, dC, sizeof(uint8_t) * HLLM, hipMemcpyDeviceToHost);

        if (verify_hll(C_ref, C, HLLM))
        {
            printf("Validated successfully!\n");
        }
        else
        {
            printf("Failed validation against NVIDIA cuBLAS.\n");
            exit(EXIT_FAILURE);
        }
    }

    // timing run(s)
    hipEventRecord(beg);
    for (int j = 0; j < REPS; j++)
    {
        // We don't reset dC between runs to save time
        runAlgo(ALGO, SIZE, HLLM, A, C, dA, dC);
        cudaCheck(hipDeviceSynchronize());
    }

    // TODO: measure timing without memory transfers?
    cudaCheck(hipEventRecord(end));
    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    float elapsed_time;
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));
    elapsed_time /= 1000.; // Convert to seconds

    double flops = SIZE;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.2f) GFLOPS. size: (%u).\n",
        elapsed_time / REPS,
        (REPS * flops * 1e-9) / elapsed_time,
        SIZE);

    // free CPU and GPU memory
    free(A);
    free(C);
    free(C_ref);
    cudaCheck(hipFree(dA));
    cudaCheck(hipFree(dC));

    return 0;
}

/** Function to check for errors in CUDA API calls */
void cudaErrorCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s: %s\n", file, line,
               hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void cublasErrorCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[CUDA ERROR] at file %s:%d:\n %s: %s\n", file, line,
               cublasGetStatusName(status), cublasGetStatusString(status));
        exit(EXIT_FAILURE);
    }
}

/** Initialize the given matrix `mat` which has `N` contiguous values. Contents of `mat` are set to random values. */
void randomize_vector(uint32_t *mat, int N, double skewness)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = distribution();
    }
}

bool verify_hll(uint8_t *expected, uint8_t *actual, int M)
{
    for (int i = 0; i < M; i++)
    {
        if (expected[i] != actual[i])
        {
            printf("Divergence! Should be %u, is %u at [%d]\n",
                    expected[i], actual[i], i);
            return false;
        }
    }
    return true;
}

uint32_t cpuHash(uint32_t data, uint32_t seed){
    uint32_t ret = data * 0x114253d5;
    ret ^= (0x2745937f * seed);
    return ret;
}

__device__  uint32_t cudaHash(uint32_t data, uint32_t seed){
    uint32_t ret = data * 0x114253d5;
    ret ^= (0x2745937f * seed);
    return ret;
}

void runCpu(int N, int M, uint32_t *A, uint8_t *C){
    for (int i = 0; i < N; ++i)
    {
        uint32_t data = A[i];
        uint32_t position = cpuHash(data, 0) % M;
        INT16_MIN value = countl_zero(cpuHash(data, 1)) + 1;
        C[position] = max(C[position], value);
    }
}

__global__ void runBasic(int N, int M, uint32_t *A, uint8_t *C){
    const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        uint32_t data = A[index];
        uint32_t position = cudaHash(data, 0) % M;
        int value = __clz(cudaHash(data, 1)) + 1;
        atomicMax(&C[position], value);
    }
}

void runAlgo(Algo algo, int N, int M, uint32_t *A, uint8_t *C, uint32_t *dA, uint8_t *dC)
{
    switch (algo)
    {
    case cpu:
        runCpu(N, M, A, C);
        break;
    case basic:
    {
        dim3 gridDim(ROUND_UP_TO_NEAREST(SIZE, BLOCK_SIZE));
        dim3 blockDim(BLOCK_SIZE);
        runBasic<<<gridDim, blockDim>>>(N, M, dA, dC);
        break;
    }
    default:
        printf("Invalid algorithm: %d\n", algo);
        exit(EXIT_FAILURE);
    }
    cudaCheck(hipDeviceSynchronize()); // wait for kernel to finish
    cudaCheck(hipGetLastError());      // check for errors from kernel run
}
